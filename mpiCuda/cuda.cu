﻿#include <hip/hip_runtime.h>


#define KERNEL_SIZE 3
#define BLOCK_SIZE  512

typedef signed int pixel_channel;

__constant__ pixel_channel kernel_cuda[KERNEL_SIZE * KERNEL_SIZE];
pixel_channel kernel_host[KERNEL_SIZE * KERNEL_SIZE];

__global__ void Pixel_Shared_Convolution(pixel_channel *channel_cuda, pixel_channel *rezult_cuda, unsigned long width, unsigned long lineQuantity)
{
  
    __shared__ pixel_channel sharedMemory [3][BLOCK_SIZE + 2];

    for(long line = 1; line < lineQuantity; line++)
    {
        long temp = blockIdx.x * BLOCK_SIZE + threadIdx.x + 1;

        sharedMemory [0][threadIdx.x+1] = channel_cuda[temp + width * (line - 1)];
        sharedMemory [1][threadIdx.x+1] = channel_cuda[temp + width * line];
        sharedMemory [2][threadIdx.x+1] = channel_cuda[temp + width * (line + 1)];

        if(threadIdx.x == 0)
        {
            temp--;
            sharedMemory [0][0] = channel_cuda[temp + width * (line-1)];
            sharedMemory [1][0] = channel_cuda[temp + width * line];
            sharedMemory [2][0] = channel_cuda[temp + width * (line+1)];
        }

        if(threadIdx.x == (BLOCK_SIZE-1))
        {
            temp += 2;
            sharedMemory [0][BLOCK_SIZE] = channel_cuda[temp + width * (line - 1)];
            sharedMemory [1][BLOCK_SIZE] = channel_cuda[temp + width * line + 2];
            sharedMemory [2][BLOCK_SIZE] = channel_cuda[temp + width * (line + 1)];
        }
        __syncthreads();

        pixel_channel Sum = 0;

        for (int i = 0; i < KERNEL_SIZE; i++)
            for (int j = 0; j < KERNEL_SIZE; j++)
                Sum += sharedMemory[j][threadIdx.x+1] * kernel_cuda[i * 3 + j];

        if (Sum < 0)
            Sum = 0;
        if (Sum > 255)
            Sum = 255;

        rezult_cuda[blockIdx.x * BLOCK_SIZE + threadIdx.x + width * line + 1] = Sum;

    }

    return;
    
}

extern "C" pixel_channel* Shared_Memory_Convolution(pixel_channel *channel, unsigned long width, unsigned long height, pixel_channel kernel[3][3], float *time)
{
  
    pixel_channel *channel_cuda, *rezult_cuda;
    pixel_channel size = width * height;

    for(int i = 0; i < 3; i++)
        for(int j = 0; j < 3; j++)
            kernel_host[i * 3 + j] = kernel[i][j];

    long block_count = 0;
    if(((width - 2)%BLOCK_SIZE) == 0)
        block_count = (width - 2)/BLOCK_SIZE;
    else
        block_count = (width - 2)/BLOCK_SIZE + 1;

    dim3 gridSize = dim3(block_count, 1, 1);
    dim3 blockSize = dim3(BLOCK_SIZE, 1, 1);

    hipEvent_t start;
    hipEvent_t stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    hipMalloc((void **)& rezult_cuda, (size + 256) * sizeof(pixel_channel));
    hipMalloc((void **)& channel_cuda, (size + 256) * sizeof(pixel_channel));

    hipMemcpy(channel_cuda, channel, size * sizeof(pixel_channel), hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(kernel_cuda), kernel_host, 9 * sizeof(pixel_channel), 0, hipMemcpyHostToDevice);

    Pixel_Shared_Convolution<<<gridSize, blockSize>>>(channel_cuda, rezult_cuda, width, (height - 2));

    hipMemcpy(channel, rezult_cuda, size * sizeof(pixel_channel), hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(time, start, stop);

    hipFree(rezult_cuda);
    hipFree(channel_cuda);

    hipDeviceReset();
    

    return channel;
}